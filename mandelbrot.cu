
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#define CUDA_KERNEL_LOOP_x(i,n) \
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; \
        i < (n); \
        i += blockDim.x * gridDim.x)

#define CUDA_KERNEL_LOOP_y(j,m) \
    for(int j = blockIdx.y * blockDim.y + threadIdx.y; \
        j < (m); \
        j += blockDim.y * gridDim.y)

__device__ int mandelbrot(hipDoubleComplex c, int threshold)
{
    hipDoubleComplex z = make_hipDoubleComplex(0, 0);
    for(int i = 0; i < threshold; i++)
    {
        z = hipCadd(hipCmul(z, z), c);
        if(hipCabs(z) > 2)
            return i;
    }
    return 0;
}

__global__ void mandelbrot_set(double xmin, double xmax, double ymin, double ymax,
                               int xn, int yn, int threshold, int *atlas)
{
    CUDA_KERNEL_LOOP_y(j, yn)
    {
        CUDA_KERNEL_LOOP_x(i, xn)
        {
            double cx = xmin + i * (xmax - xmin) / xn;
            double cy = ymin + j * (ymax - ymin) / yn;
            hipDoubleComplex c = make_hipDoubleComplex(cx, cy);
            atlas[j * xn + i] = mandelbrot(c, threshold);
        }
    }
}
